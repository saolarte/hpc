
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#define BLOCK_SIZE 1024
#define MAX_MASK_WIDTH 5
#define TILE_SIZE 1024
__constant__ int M[MAX_MASK_WIDTH];
using namespace std;



void convolutionBasic(int *N,int *M,int *P,int Mask_Width,int Width){
for(int i=0;i<Width;i++){
int N_start_point = i - (Mask_Width/2);
int Pvalue=0;
for (int j= 0;j<Mask_Width;j++) {
if (N_start_point+j >= 0 && N_start_point + j < Width) {
Pvalue+=N[N_start_point+j]*M[j];
}
}
P[i]=Pvalue;
}
}




__global__ void KernelConvolutionBasic(int *N,int *M,int *P,int Mask_Width,int Width){
int i = blockIdx.x*blockDim.x + threadIdx.x;
int N_start_point = i - (Mask_Width/2);
int Pvalue=0;
for (int j= 0;j<Mask_Width;j++) {
if (N_start_point+j >= 0 && N_start_point + j < Width) {
Pvalue+=N[N_start_point+j]*M[j];
}
}
if(i<Width)
P[i]=Pvalue;
}







__global__ void KernelConvolutionTile(int *N, int *P, int Mask_Width,int Width) {
int i = blockIdx.x*blockDim.x + threadIdx.x;
__shared__ int N_ds[TILE_SIZE + MAX_MASK_WIDTH -1];
int n = Mask_Width/2;
int halo_index_left = (blockIdx.x - 1)*blockDim.x + threadIdx.x;
if (threadIdx.x >= blockDim.x - n) {
N_ds[threadIdx.x - (blockDim.x - n)] =(halo_index_left < 0) ? 0 : N[halo_index_left];
}
if(i<Width)
N_ds[n + threadIdx.x] = N[i];
else
N_ds[n + threadIdx.x] = 0;
int halo_index_right = (blockIdx.x + 1)*blockDim.x + threadIdx.x;
if (threadIdx.x < n) {
N_ds[n + blockDim.x + threadIdx.x] =(halo_index_right >= Width) ? 0 : N[halo_index_right];
}
__syncthreads();
int Pvalue = 0;
for(int j = 0; j < Mask_Width; j++) {
Pvalue += N_ds[threadIdx.x + j]*M[j];
}
P[i] = Pvalue;
}


__global__ void KernelConvolutionCaching(int *N,int *P,int Mask_Width,int Width){
int i = blockIdx.x*blockDim.x + threadIdx.x;
int N_start_point = i - (Mask_Width/2);
int Pvalue=0;
for (int j= 0;j<Mask_Width;j++) {
if (N_start_point+j >= 0 && N_start_point + j < Width) {
Pvalue+=N[N_start_point+j]*M[j];
}
}
P[i]=Pvalue;
}


void imprimirVec(int *V,int n){
cout<<"|";
for(int i=0;i<n;i++)
cout<<V[i]<<"|";
cout<<endl;
}
void llenar(int *V,int N,int flag){
if(flag==1)
for(int i = 1; i <=N; i++ )
V[i-1] = i;
else
for(int i = 1; i <=N; i++ )
V[i-1] = 0;
}
void compare(int*A,int *B1,int *B2,int *B3,int width){
for(int i=0;i<width;i++)
if(((A[i]!=B1[i])||(A[i]!=B2[i]))||(A[i]!=B3[i])){
cout<<"Los vectores no son iguales"<<endl;
return;
}
cout<<"Los vectores son iguales"<<endl;
}



int main(){
int N=1000;
int bytes=(N)*sizeof(int);
int bytesM=MAX_MASK_WIDTH *sizeof(int);
int *V=(int*)malloc(bytes);
int *P=(int*)malloc(bytes);
int Mask[MAX_MASK_WIDTH]={3,4,5,4,3};
llenar(V,N,1);
llenar(P,N,0);
//Convolucion secuencial
clock_t start = clock();
convolutionBasic(V,Mask,P,5,N);
clock_t end= clock();
double elapsed_seconds=end-start;
printf("Tiempo transcurrido Secuencial: %lf\n", (elapsed_seconds / CLOCKS_PER_SEC));
//imprimirVec(P,N);
/////////////////////////
//Bloque y grid
float blocksize=BLOCK_SIZE;
dim3 dimGrid(ceil(N/blocksize),1,1);
dim3 dimBlock(blocksize,1,1);
//////////////////////Convolucion Paralelo Basic////////////////////////
//Variables para el kernel
int *d_V1;
int *d_P1;
int *d_Mask1;
int *P_out1=(int*)malloc(bytes);
int *P_in1=(int*)malloc(bytes);
llenar(P_in1,N,0);
hipMalloc(&d_V1,bytes);
hipMalloc(&d_P1,bytes);
hipMalloc(&d_Mask1,bytesM);
start=clock();
hipMemcpy(d_V1, V, bytes, hipMemcpyHostToDevice);
hipMemcpy(d_P1, P_in1, bytes, hipMemcpyHostToDevice);
hipMemcpy(d_Mask1, Mask, bytesM, hipMemcpyHostToDevice);
KernelConvolutionBasic<<<dimGrid,dimBlock>>>(d_V1,d_Mask1,d_P1,MAX_MASK_WIDTH,N);
hipDeviceSynchronize();
hipMemcpy(P_out1,d_P1, bytes, hipMemcpyDeviceToHost );
end=clock();
double elapsed_seconds1=end-start;
printf("Tiempo Paralelo Basic: %lf\n", (elapsed_seconds1 / CLOCKS_PER_SEC));
//imprimirVec(P_out1,N);
cout<<"Aceleracion Secuencial Vs. Paralela básica: "<<elapsed_seconds/elapsed_seconds1<<endl<<endl;
free(P_in1);
hipFree(d_V1);
hipFree(d_P1);
//////////////////////////////////////////////////////////////////////////
/////////////////////Convolucion Paralelo Caching/////////////////////////
int *d_V2;
int *d_P2;
int *P_out2=(int*)malloc(bytes);
int *P_in2=(int*)malloc(bytes);
//Constant Memory
hipMemcpyToSymbol(HIP_SYMBOL(M),Mask,bytesM);
llenar(P_in2,N,0);
hipMalloc(&d_V2,bytes);
hipMalloc(&d_P2,bytes);
start=clock();
hipMemcpy(d_V2, V, bytes, hipMemcpyHostToDevice);
hipMemcpy(d_P2, P_in2, bytes, hipMemcpyHostToDevice);
KernelConvolutionCaching<<<dimGrid,dimBlock>>>(d_V2,d_P2,MAX_MASK_WIDTH,N);
hipDeviceSynchronize();
hipMemcpy(P_out2,d_P2, bytes, hipMemcpyDeviceToHost );
end=clock();
double elapsed_seconds2=end-start;
printf("Tiempo transcurrido Paralelo Caching: %lf\n", (elapsed_seconds2 / CLOCKS_PER_SEC));
//imprimirVec(P_out2,N);
cout<<"Aceleracion Secuencial Vs. Paralelo con Caching: "<<elapsed_seconds/elapsed_seconds2<<endl<<endl;
free(P_in2);
hipFree(d_V2);
hipFree(d_P2);
/////////////////////////////////////////////////////////////////////////////
//////////////////////////Convolucion Paralelo Tile///////////////////////////
//Variables para el kernel
int *d_V3;
int *d_P3;
int *P_out3=(int*)malloc(bytes);
int *P_in3=(int*)malloc(bytes);
llenar(P_in3,N,0);
hipMalloc(&d_V3,bytes);
hipMalloc(&d_P3,bytes);
start=clock();
hipMemcpy(d_V3, V, bytes, hipMemcpyHostToDevice);
hipMemcpy(d_P3, P_in3, bytes, hipMemcpyHostToDevice);
KernelConvolutionTile<<<dimGrid,dimBlock>>>(d_V3,d_P3,MAX_MASK_WIDTH,N);
hipDeviceSynchronize();
hipMemcpy(P_out3,d_P3, bytes, hipMemcpyDeviceToHost );
end=clock();
double elapsed_seconds3=end-start;
printf("Tiempo transcurrido Paralelo Tile: %lf\n", (elapsed_seconds3 / CLOCKS_PER_SEC));
//imprimirVec(P_out3,N);
cout<<"Aceleracion Secuencial Vs. Paralelo Tile: "<<elapsed_seconds/elapsed_seconds3<<endl;
free(P_in3);
hipFree(d_V3);
hipFree(d_P3);
/////////////////////////////////////////////////////////////////////////////
compare(P,P_out1,P_out2,P_out3,N);
free(V);
free(P);
free(P_out1);
free(P_out2);
free(P_out3);
//free(Mask);
return 0;
}
