#include <hip/hip_runtime.h>
#include <stdio.h>
#include<time.h>








///////////////////////////////////////////FUNCION INICIALIZAR  LAS MATRICES//////////////////////////////////
//////////////////////////Todas la posiciones de la matriz son inicializadas en 1/////////////////////////////
void inic_matriz(int* matriz, int tam){
  for(int i=0; i<tam*tam; i++){
    matriz[i] = 1;
  }

}



/////////////////////////////////////MULTIPLICACION DE MATRICES SECUENCIAL///////////////////////////////////
int mult_matrices(int* A, int* B, int* C, int tam){
  int acum=0;
  for (int fila=0; fila<tam; ++fila){
    for (int col=0;col<tam; ++col){
      acum=0;
      for (int k=0; k<tam; ++k){
        acum += A[fila*col + k]*B[k*tam+col]; 
        
      }
     C[fila*fila+col]=acum;
      
      
    }
  
  
  }
  return 0;
    
  
}

////////////////////////////KERNEL MULTIPLICACION DE MATRICES SIN TILING//////////////////////////
__global__ void matrixMulKernel(int *d_A, int *d_B, int *d_C, int tam){
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;
    int Pvalue;
    if((row < tam)&&(col < tam)){
        Pvalue = 0;
        for (int k = 0; k < tam ; ++k){
            Pvalue += d_A[row*tam+k] * d_B[k*tam+col];
        }
        d_C[row*tam+col] = Pvalue;
    }
}






int main(){
  clock_t sec_ini, sec_fin, par_ini, par_fin;
  double tiempo_sec, tiempo_par;
   
  int tam= 1024;
  int size= tam*tam*sizeof (int);
  
  
  
  //////////////////////////////VARIABLES EN HOST/////////////////////////////////
  int *h_A, *h_B, *h_C, *sec_C;
  ////////////////////////////////Reservar memoria///////////////////////////////
  h_A = (int * ) malloc (size);
  h_B = (int * ) malloc (size);
  h_C = (int * ) malloc (size);
  sec_C = (int * ) malloc (size);
  /////////////////////////Inicializar variables en host/////////////////////////
  inic_matriz(h_A, tam);
  inic_matriz(h_B, tam);
  inic_matriz(h_C, tam);
  ////////////////////////////VARIABLES EN HOST////////////////////////////////
   
  
  ///////////////////////////////REALIZAR MULTIPLICACION SECUENCIAL////////////////////////////
  
  sec_ini=clock();
  
  mult_matrices(h_A,h_B,sec_C, tam);
  
  sec_fin= clock();
  tiempo_sec= ((double) (sec_fin - sec_ini)) / CLOCKS_PER_SEC;
  printf("EL ALGORITMO SECUENCIAL TARDO: %.10f\n", tiempo_sec);
//////////////////////////////////////////////////////////////////////////////////////////// 
  
   
  
  
/////////////////////////////EJECUCUCION ALGORITMO PARALELO/////////////////////////////////////////
 /////////////////////////////Variables en device////////////////////////////
  int *d_A, *d_B, *d_C;
///////////////////////////////Reserva de memoria////////////////////////////////
  
  hipMalloc((void**)&d_A,size);
  hipMalloc((void**)&d_B,size);
  hipMalloc((void**)&d_C,size);

  
  par_ini=clock();
  
  
  hipMemcpy(d_A,h_A,size,  hipMemcpyHostToDevice);
  hipMemcpy(d_B,h_B,size,  hipMemcpyHostToDevice);
  hipMemcpy(d_C,h_C,size,  hipMemcpyHostToDevice);
  
  
  int blockSize = 4;
  dim3 dimBlock(blockSize,blockSize,1);
  dim3 dimGrid(ceil(tam/float(blockSize)),ceil(tam/float(blockSize)),1);
  matrixMulKernel<<<dimGrid,dimBlock>>>(d_A,d_B,d_C,tam);
  hipDeviceSynchronize();
  hipMemcpy(h_C,d_C,size,hipMemcpyDeviceToHost);
   
  
  par_fin=clock();
  tiempo_par= ((double) (par_fin - par_ini)) / CLOCKS_PER_SEC;
  printf("EL ALGORITMO PARALELO TARDO: %.10f\n", tiempo_par);
    
  
  hipFree(d_A); hipFree(d_B);  hipFree(d_C);
  
}


